#include <hip/hip_runtime.h>
#include <iostream>

template<class T>
__device__ T plus_scan(T *x)
{
	unsigned int i = threadIdx.x;
	unsigned int n = blockDim.x;
	for(unsigned int offset=1; offset<n; offset *= 2)
	{
		T t;
		if(i>=offset) t = x[i-offset];
		__syncthreads();
		if(i>=offset) x[i] = t + x[i];
		__syncthreads();
	}
	return x[i];
}

__device__ void partition_by_bit(unsigned int *values, unsigned int bit)
{
	unsigned int i = threadIdx.x;
	unsigned int size = blockDim.x;
	unsigned int x_i = values[i];
	unsigned int p_i = (x_i >> bit) & 1;
	values[i] = p_i;
	__syncthreads();
	// Compute number of T bits up to and including p_i.
	// Record the total number of F bits as well.
	unsigned int T_before = plus_scan(values);
	unsigned int T_total = values[size-1];
	unsigned int F_total = size - T_total;
	__syncthreads();
	// Write every x_i to its proper place
	if( p_i )
	values[T_before-1 + F_total] = x_i;
	else
	values[i - T_before] = x_i;
}

__global__ void radix_sort(unsigned int *values)
{
	for(int bit=0; bit<32; ++bit)
	{
	partition_by_bit(values, bit);
	__syncthreads();
	}
}

int main() {
    // Number of elements to sort
    int arraySize;
	printf("Provide the size of the array: " );
    scanf("%d",&arraySize);
    unsigned int *h_values = new unsigned int[arraySize]; // host array
    unsigned int *d_values; // device array

    // Initialize host array with data
    for(int i = 0; i < arraySize; ++i) {
		printf("Provide the values for array's position %d: \n",i);
		scanf("%u",&h_values[i]);
    }

    // Allocate memory on the device
    hipMalloc((void**)&d_values, arraySize * sizeof(unsigned int));

    // Copy data from host to device
    hipMemcpy(d_values, h_values, arraySize * sizeof(unsigned int), hipMemcpyHostToDevice);

    // Launch the kernel
	int threadsPerBlock = 256;
    int nblocks = (arraySize + threadsPerBlock - 1) / threadsPerBlock; // This ensures rounding up if n is not a multiple of threadsPerBlock
    radix_sort<<<nblocks, arraySize>>>(d_values);

    // Copy sorted array back to host
    hipMemcpy(h_values, d_values, arraySize * sizeof(unsigned int), hipMemcpyDeviceToHost);

    // Display sorted data
	printf("\nSorted Array: ");
    for (int i = 0; i < arraySize; ++i) {
        printf("%d ", h_values[i]);
    }
    printf("\n");

    // Free device memory
    hipFree(d_values);

    // Free host memory
    delete[] h_values;

    return 0;
}